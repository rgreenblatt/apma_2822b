
#include <hip/hip_runtime.h>
#include <algorithm>
#include <assert.h>
#include <cfloat>
#include <cstdint>
#include <iostream>
#include <omp.h>
#include <random>
#include <stdio.h>
#include <stdlib.h>
#include <utility>
#include <vector>

#define cuda_error_chk(ans)                                                    \
  { cuda_assert((ans), __FILE__, __LINE__); }
inline void cuda_assert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    exit(code);
  }
}

const unsigned FULL_MASK = 0xffffffff;

// primarily for camel case avoidance
const unsigned WARP_SIZE = 32;
const unsigned LOG_WARP_SIZE = 5;

template <typename T>
__forceinline__ __device__ T warp_prefix_sum(T val, T *sum,
                                             unsigned thread_idx) {
  T orig_val = val;

  for (uint8_t i = 1; i < WARP_SIZE; i <<= 1) {
    auto adder = __shfl_up_sync(FULL_MASK, val, i);

    if (thread_idx % WARP_SIZE >= i) {
      val += adder;
    }
  }

  if (sum != NULL) {
    *sum = val;
  }

  return val - orig_val;
}

const uint8_t BITS_PER_PASS = 2;
const uint8_t BINS_PER_PASS = 4;

union packed {
  uint32_t bytes;
  uint8_t vals[BINS_PER_PASS];
};

__global__ void maxes(uint32_t **data, uint32_t **max_vals, uint32_t **max_locs,
                      uint32_t m, u_int32_t nth, uint32_t num_warps_per_n) {

  extern __shared__ uint32_t shared_memory[];

  uint32_t *bin_sums = shared_memory;

  size_t current_size = num_warps_per_n * BINS_PER_PASS;
  uint32_t *working_data = &bin_sums[current_size];

  current_size += m;
  uint32_t *sorted_data = &bin_sums[current_size];

  current_size += m;
  uint32_t *indexes = &bin_sums[current_size];

  current_size += m;
  uint32_t *sorted_indexes = &bin_sums[current_size];

  current_size += m;
  uint32_t *n_maxes = &bin_sums[current_size];

  current_size += BINS_PER_PASS;
  packed *warp_maxes = (packed *)&bin_sums[current_size];

  current_size += num_warps_per_n;

  uint32_t m_per_warp = m / num_warps_per_n;
  uint32_t m_iterations = (m_per_warp - 1 + WARP_SIZE) / WARP_SIZE;

  packed *warp_iteration_maxes = (packed *)&bin_sums[current_size];
  current_size += num_warps_per_n * m_iterations;

  size_t n_idx = blockIdx.x;
  uint32_t m_index = threadIdx.x;
  unsigned warp_idx = threadIdx.x >> LOG_WARP_SIZE;

  // it is possible to change this so that the extra values are distributed
  // evenly over warps
  uint32_t m_local_index = m_index % WARP_SIZE;
  uint32_t m_offset = m_per_warp * warp_idx;
  uint32_t m_start = m_local_index + m_offset;
  uint32_t m_end =
      (warp_idx == num_warps_per_n - 1) ? m : m_per_warp * (warp_idx + 1);

  for (uint32_t j = m_start; j < m_end; j += WARP_SIZE) {
    working_data[j] = data[n_idx][j];
    indexes[j] = j;
  }

  bool is_last_thread_in_warp = (m_index + 1) % WARP_SIZE == 0;

  packed *counts = new packed[m_iterations]();

  for (uint8_t shift = 0; shift < 8 * sizeof(uint32_t);
       shift += BITS_PER_PASS) {

    for (uint32_t i = 0; i < m_iterations; ++i) {
      counts[i].bytes = 0;
    }

    for (size_t j = m_start; j < m_end; j += WARP_SIZE) {
      counts[(j - m_offset) / WARP_SIZE]
          .vals[(working_data[j] >> shift) & (BINS_PER_PASS - 1)]++;
      counts[(j - m_offset) / WARP_SIZE].bytes = warp_prefix_sum(
          counts[(j - m_offset) / WARP_SIZE].bytes,
          (is_last_thread_in_warp || j == m_end - 1)
              ? &warp_iteration_maxes[warp_idx * m_iterations +
                                      (j - m_offset) / WARP_SIZE]
                     .bytes
              : nullptr,
          threadIdx.x);
    }

    if (m_local_index < m_iterations) {
      warp_iteration_maxes[warp_idx * m_iterations + m_local_index]
          .bytes = warp_prefix_sum(
          warp_iteration_maxes[warp_idx * m_iterations + m_local_index].bytes,
          (m_local_index == m_iterations - 1) ? &warp_maxes[warp_idx].bytes
                                              : nullptr,
          threadIdx.x);
    }

    if (is_last_thread_in_warp) {
      for (uint32_t bin = 0; bin < BINS_PER_PASS; bin++) {
        bin_sums[BINS_PER_PASS * warp_idx + bin] =
            warp_maxes[warp_idx].vals[bin];
        /* printf("warp_idx: %u, warp max: %u\n", warp_idx, */
        /*        warp_maxes[warp_idx].array_int[bin]); */
      }
    }

    __syncthreads();

    // assumption is that num_warps_per_n is less than the WARP_SIZE
    if (m_index < num_warps_per_n) {
      for (uint8_t bin = 0; bin < BINS_PER_PASS; bin++) {
        bin_sums[BINS_PER_PASS * m_index + bin] = warp_prefix_sum(
            bin_sums[BINS_PER_PASS * m_index + bin],
            (m_index == num_warps_per_n - 1) ? &n_maxes[bin] : (uint32_t *)0,
            threadIdx.x);
      }
    }

    if (m_index < BINS_PER_PASS) {
      n_maxes[m_index] =
          warp_prefix_sum(n_maxes[m_index], (uint32_t *)0, threadIdx.x);
    }

    __syncthreads();

    for (size_t j = m_start; j < m_end; j += WARP_SIZE) {
      uint8_t bin = (working_data[j] >> shift) & (BINS_PER_PASS - 1);
      uint32_t idx = counts[(j - m_offset) / WARP_SIZE].vals[bin] +
                     bin_sums[BINS_PER_PASS * warp_idx + bin] + n_maxes[bin] +
                     warp_iteration_maxes[warp_idx * m_iterations +
                                          (j - m_offset) / WARP_SIZE]
                         .vals[bin];

      sorted_data[idx] = working_data[j];
      sorted_indexes[idx] = indexes[j];
    }

    __syncthreads();

    uint32_t *temp_ptr = working_data;
    working_data = sorted_data;
    sorted_data = temp_ptr;

    temp_ptr = indexes;
    indexes = sorted_indexes;
    sorted_indexes = temp_ptr;
  }

  if (m_start == 0) {
    max_locs[n_idx][0] = indexes[0];
    max_vals[n_idx][0] = sorted_data[0];
  }
  if (nth >= m_start && nth < m_end) {
    max_locs[n_idx][1] = indexes[nth];
    max_vals[n_idx][1] = sorted_data[nth];
  }

  delete[] counts;
}

int main() {
  int ngpus = 0;
  hipGetDeviceCount(&ngpus);
  printf("ngpus = %d\n", ngpus);
  if (ngpus > 0)
    hipSetDevice(0);
  else
    return 0;

  uint32_t m = 1024;
  uint32_t n = 16384;

  std::vector<uint32_t> range_to_m;

  for (uint32_t i = 0; i < m; ++i) {
    range_to_m.push_back(i);
  }

  uint32_t **data;
  uint32_t **max_vals;
  uint32_t **max_locs;

  /* #ifdef USE_NVTX */
  /*   // nvtxRangePushA("A"); */
  /*   nvtxRangeId_t nvtx_1 = nvtxRangeStartA("A"); */
  /* #endif */

  hipMallocManaged(&data, n * sizeof(uint32_t *));
  hipMallocManaged(&max_vals, n * sizeof(uint32_t *));
  hipMallocManaged(&max_locs, n * sizeof(size_t *));

  hipMallocManaged(&data[0], n * m * sizeof(uint32_t));
  hipMallocManaged(&max_vals[0], n * m * sizeof(uint32_t));
  hipMallocManaged(&max_locs[0], n * m * sizeof(size_t));

  for (size_t i = 1; i < n; ++i) {
    data[i] = data[0] + i * m;
    max_vals[i] = max_vals[0] + i * m;
    max_locs[i] = max_locs[0] + i * m;
  }

  /* #ifdef USE_NVTX */
  /*   nvtxRangeEnd(nvtx_1); */
  /*   //nvtxRangePop(); */
  /* #endif */

  /* std::random_device r; */
  /* std::srand(r()); */

  for (unsigned i = 0; i < n; ++i) {
    std::random_shuffle(range_to_m.begin(), range_to_m.end());
    for (unsigned j = 0; j < m; ++j) {
      data[i][j] = range_to_m[j];
    }
  }

  // set to obvious failure
  for (size_t i = 0; i < n; ++i) {
    for (size_t j = 0; j < 2; ++j) {
      max_locs[i][j] = static_cast<uint32_t>(-1);
      max_vals[i][j] = static_cast<uint32_t>(-1);
    }
  }

  uint32_t nth_max = static_cast<uint32_t>(std::rand()) % m;
  std::cout << "nth is " << nth_max << std::endl;

  /* ---------------  TASK 1  ------------ */

  /* for (size_t i = 0; i < n; ++i) { */
  /*   assert(max_vals[i][0] == 0); */
  /*   assert(max_vals[i][1] == nth_max); */
  /* } */

  /* std::cout << "==== cpu passed tests ====" << std::endl; */

  /* ---------------  TASK 2  ------------ */

  /* ---------------  TASK 3  ------------ */

  // write GPU code to find the maximum in each row of data, i.e  MAX(data[i])
  // for each i also find the locaiton of each maximum

  // write GPU code to find the first maximum and the Nth maximum value in each
  // row of data, i.e  MAX(data[i]) for each i also find the locaiton of each
  // maximum

  // set to obvious failure
  for (size_t i = 0; i < n; ++i) {
    for (size_t j = 0; j < 2; ++j) {
      max_locs[i][j] = static_cast<uint32_t>(-1);
      max_vals[i][j] = static_cast<uint32_t>(-1);
    }
  }

  uint32_t num_threads_per_block = 256;
  uint32_t num_warps_per_n =
      std::min((num_threads_per_block + WARP_SIZE - 1) / WARP_SIZE,
               ((m + WARP_SIZE - 1) / WARP_SIZE));

  uint32_t m_per_warp = m / num_warps_per_n;
  uint32_t m_iterations = (m_per_warp - 1 + WARP_SIZE) / WARP_SIZE;

  // assumptions:
  // - values per warp < 256
  // - num_warps_per_n is less than the WARP_SIZE
  // - num theads is a multiple of the warp size

  assert(m_per_warp < static_cast<uint8_t>(-1));
  assert(num_threads_per_block / WARP_SIZE < WARP_SIZE);
  assert(num_threads_per_block % WARP_SIZE == 0);

  size_t shared_count = num_warps_per_n * BINS_PER_PASS + num_warps_per_n +
                        +num_warps_per_n * m_iterations + m * 4 + BINS_PER_PASS;

  maxes<<<n, num_threads_per_block, shared_count * sizeof(int32_t)>>>(
      data, max_vals, max_locs, m, nth_max, num_warps_per_n);

  cuda_error_chk(hipDeviceSynchronize());

  for (size_t i = 0; i < n; ++i) {
    assert(max_vals[i][0] == 0);
    assert(max_vals[i][1] == nth_max);
  }

  std::cout << "==== gpu passed tests ====" << std::endl;

  // print results;
  hipDeviceSynchronize();

  hipFree(data[0]);
  hipFree(data);

  return 0;
}
