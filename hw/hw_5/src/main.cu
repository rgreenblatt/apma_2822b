#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <cstdint>
#include <iostream>
#include <omp.h>
#include <stdio.h>
#include <utility>
#include <vector>
#include <stdlib.h>

#define cuda_error_chk(ans)                                                    \
  { cuda_assert((ans), __FILE__, __LINE__); }
inline void cuda_assert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    exit(code);
  }
}

#define USE_NVTX

#ifdef USE_NVTX
#include "nvToolsExt.h"
#endif

#define FULL_MASK 0xffffffff

// primarily for camel case avoidance
enum : uint32_t { warp_size = 32, log_warp_size = 5 };

template <typename T>
__forceinline__ __device__ T warp_prefix_sum(T val, T *sum,
                                             uint32_t thread_idx) {
  T orig_val = val;

  for (uint8_t i = 1; i < warp_size; i <<= 1) {
    auto adder = __shfl_up_sync(FULL_MASK, val, i);

    if (thread_idx % warp_size >= i) {
      val += adder;
    }
  }

  if (sum != NULL) {
    *sum = val;
  }

  return val - orig_val;
}

__global__ void test_prefix_sum(uint32_t *data, uint32_t *returned,
                                uint32_t n) {
  uint32_t sum;
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {
    returned[index] = warp_prefix_sum(data[index], &sum, threadIdx.x);
    printf("adding index: %lu, as: %u, sum: %u\n", index, returned[index], sum);
  }
}

#define BITS_PER_PASS 2
#define BINS_PER_PASS 4

union packed
{
  uint32_t packed_int;
  uint8_t array_int[BINS_PER_PASS];
};

__global__ void maxes(uint32_t **data, uint32_t **max_vals, uint32_t **max_locs,
                      uint32_t n, uint32_t m, u_int32_t nth,
                      uint32_t total_count,
                      uint32_t num_warps_per_n) {

  extern __shared__ uint32_t shared_memory[];

  uint32_t *bin_sums = shared_memory;
  size_t size_bins = n * num_warps_per_n * BINS_PER_PASS;

  size_t current_size = size_bins;
  uint32_t *working_data = &bin_sums[current_size];

  size_t size_data = n * m;
  current_size += size_data;
  uint32_t *sorted_data = &bin_sums[current_size];

  current_size += size_data;
  uint32_t *indexes = &bin_sums[current_size];

  current_size += size_data;
  uint32_t *sorted_indexes = &bin_sums[current_size];

  current_size += size_data;
  uint32_t *n_maxes = &bin_sums[current_size];

  current_size += BINS_PER_PASS * n;
  packed *warp_maxes = (packed *)&bin_sums[current_size];

  size_t size_bins_warp = n * num_warps_per_n;
  current_size += size_bins_warp;

  uint32_t m_per_warp = m / num_warps_per_n;  
  uint32_t m_iterations = (m_per_warp - 1 + warp_size) / warp_size;

  packed *warp_iteration_maxes = (packed *)&bin_sums[current_size];
  current_size += size_bins_warp * m_iterations;

  size_t index = blockIdx.x * blockDim.x + threadIdx.x;

  size_t n_start = (index * n) / total_count;
  size_t n_stop_iter = ullmin(((index + 1) * n) / total_count + 1, n);

  uint32_t m_index = static_cast<uint32_t>(index - (n_start * total_count) / n);

  auto warp_id = threadIdx.x >> log_warp_size;

  uint32_t m_local_index = m_index % warp_size;

  uint32_t m_offset = m_per_warp * warp_id;
  uint32_t m_start = m_local_index + m_offset;
  uint32_t m_end =
      (warp_id == num_warps_per_n - 1) ? m : m_per_warp * (warp_id + 1);

  for (size_t i = n_start; i < n_stop_iter; ++i) {
    for (uint32_t j = m_start; j < m_end; j += warp_size) {
      indexes[i * m + j] = j;
      working_data[i * m + j] = data[i][j];
    }
  }

  __syncthreads();

  bool is_last_thread_in_warp = (m_index + 1) % warp_size == 0;

  for (uint8_t shift = 0; shift < 8 * sizeof(uint32_t); shift += BITS_PER_PASS) {
    for (size_t i = n_start; i < n_stop_iter; ++i) {

#define INDEX_BIN_ARRAY(arr, iter_index)                                       \
  (arr[i * num_warps_per_n * BINS_PER_PASS + iter_index * BINS_PER_PASS + bin])

      auto warp_idx = (threadIdx.x >> log_warp_size) + i * num_warps_per_n;

      packed *counts = new packed[m_iterations]();

      for (size_t j = m_start; j < m_end; j += warp_size) {
        counts[(j - m_offset) / warp_size].array_int[(working_data[i * m + j] >> shift) &
                                        (BINS_PER_PASS - 1)]++;
        counts[(j - m_offset) / warp_size].packed_int = warp_prefix_sum(
            counts[(j - m_offset) / warp_size].packed_int,
            (is_last_thread_in_warp || j == m_end - 1)
                ? &warp_iteration_maxes[warp_idx * m_iterations + (j - m_offset) / warp_size]
                       .packed_int
                : nullptr,
            threadIdx.x);
      }

      if (m_local_index < m_iterations) {
        warp_iteration_maxes[warp_idx * m_iterations + m_local_index]
            .packed_int = warp_prefix_sum(
            warp_iteration_maxes[warp_idx * m_iterations + m_local_index]
                .packed_int,
            (m_local_index == m_iterations - 1)
                ? &warp_maxes[warp_idx].packed_int
                : nullptr,
            threadIdx.x);
      }

      if (is_last_thread_in_warp) {
        for (uint32_t bin = 0; bin < BINS_PER_PASS; bin++) {
          INDEX_BIN_ARRAY(bin_sums, warp_id) =
              warp_maxes[warp_idx].array_int[bin];
        }
      }

    __syncthreads();

    // assumption is that num_warps_per_n is less than the warp_size
    if (m_index < num_warps_per_n) {
      for (uint8_t bin = 0; bin < BINS_PER_PASS; bin++) {
        INDEX_BIN_ARRAY(bin_sums, m_index) = warp_prefix_sum(
            INDEX_BIN_ARRAY(bin_sums, m_index),
            (m_index == num_warps_per_n - 1) ? &n_maxes[i * BINS_PER_PASS + bin]
                                             : (uint32_t *)0,
            threadIdx.x);
      }
    }

   if (m_index < BINS_PER_PASS) {
      n_maxes[i * BINS_PER_PASS + m_index] = warp_prefix_sum(
          n_maxes[i * BINS_PER_PASS + m_index], (uint32_t *)0, threadIdx.x);
    }
    __syncthreads();

    for (size_t j = m_start; j < m_end; j += warp_size) {
      uint8_t bin = (working_data[i * m + j] >> shift) & (BINS_PER_PASS - 1);
      uint32_t idx =
          counts[(j - m_offset) / warp_size].array_int[bin] +
          INDEX_BIN_ARRAY(bin_sums, warp_id) +
          n_maxes[i * BINS_PER_PASS + bin] +
          warp_iteration_maxes[warp_idx * m_iterations + (j - m_offset) / warp_size]
              .array_int[bin];

      sorted_data[i * m + idx] = working_data[i * m + j];
      sorted_indexes[i * m + idx] = indexes[i * m + j];
    }

    __syncthreads();

    uint32_t *temp_ptr = working_data;
    working_data = sorted_data;
    sorted_data = temp_ptr;

    temp_ptr = indexes;
    indexes = sorted_indexes;
    sorted_indexes = temp_ptr;



#undef INDEX_BIN_ARRAY
#undef BIN_LOOP
      }
  }


  for (size_t i = n_start; i < n_stop_iter; ++i) {
    if (m_start == 0) {
      max_locs[n_start][0] = indexes[i * m];
      max_vals[n_start][0] = sorted_data[i * m];
    }
    if (nth >= m_start && nth < m_end) {
      max_locs[n_start][1] = indexes[nth + i * m];
      max_vals[n_start][1] = sorted_data[nth + i * m];
    }
  }
}

int main() {
  int ngpus = 0;
  hipGetDeviceCount(&ngpus);
  printf("ngpus = %d\n", ngpus);
  if (ngpus > 0)
    hipSetDevice(0);
  else
    return 0;

  uint32_t m = 1024;
  uint32_t n = 1;

  std::vector<uint32_t> range_to_m;

  for (uint32_t i = 0; i < m; ++i) {
    range_to_m.push_back(i);
  }

  uint32_t **data;
  uint32_t **max_vals;
  uint32_t **max_locs;

  /* #ifdef USE_NVTX */
  /*   // nvtxRangePushA("A"); */
  /*   nvtxRangeId_t nvtx_1 = nvtxRangeStartA("A"); */
  /* #endif */

  hipMallocManaged(&data, n * sizeof(uint32_t *));
  hipMallocManaged(&max_vals, n * sizeof(uint32_t *));
  hipMallocManaged(&max_locs, n * sizeof(size_t *));

  hipMallocManaged(&data[0], n * m * sizeof(uint32_t));
  hipMallocManaged(&max_vals[0], n * m * sizeof(uint32_t));
  hipMallocManaged(&max_locs[0], n * m * sizeof(size_t));

  for (size_t i = 1; i < n; ++i) {
    data[i] = data[0] + i * m;
    max_vals[i] = max_vals[0] + i * m;
    max_locs[i] = max_locs[0] + i * m;
  }

  /* #ifdef USE_NVTX */
  /*   nvtxRangeEnd(nvtx_1); */
  /*   //nvtxRangePop(); */
  /* #endif */

  for (unsigned i = 0; i < n; ++i) {
    std::random_shuffle(range_to_m.begin(), range_to_m.end());
    for (unsigned j = 0; j < m; ++j) {
      data[i][j] = range_to_m[j];
    }
  }

  // set to obvious failure
  for (size_t i = 0; i < n; ++i) {
    for (size_t j = 0; j < 2; ++j) {
      max_locs[i][j] = static_cast<uint32_t>(-1);
      max_vals[i][j] = static_cast<uint32_t>(-1);
    }
  }

  uint32_t nth_max = static_cast<uint32_t>(std::rand()) % m;
  std::cout << "nth is " << nth_max << std::endl;

  /* ---------------  TASK 1  ------------ */

  std::cout << "==== cpu ====\n";
  for (size_t i = 0; i < n; i++) {
    std::cout << "value: " << max_vals[i][0] << " loc: " << max_locs[i][0]
              << " value nth: " << max_vals[i][1]
              << " loc nth: " << max_locs[i][1] << "\n";
  }
  std::cout << std::endl;

  /* ---------------  TASK 2  ------------ */

  /* ---------------  TASK 3  ------------ */

  // write GPU code to find the maximum in each row of data, i.e  MAX(data[i])
  // for each i also find the locaiton of each maximum

  // write GPU code to find the first maximum and the Nth maximum value in each
  // row of data, i.e  MAX(data[i]) for each i also find the locaiton of each
  // maximum

  // set to obvious failure
  for (size_t i = 0; i < n; ++i) {
    for (size_t j = 0; j < 2; ++j) {
      max_locs[i][j] = static_cast<uint32_t>(-1);
      max_vals[i][j] = static_cast<uint32_t>(-1);
    }
  }

  //test code
  /* const uint test_size = 16; */
  /* uint32_t *data_test; */
  /* uint32_t *return_test; */
  /* hipMallocManaged(&data_test, n * sizeof(uint32_t)); */
  /* hipMallocManaged(&return_test, n * sizeof(uint32_t)); */

  /* for (size_t i = 0; i < test_size; ++i) { */
  /*   return_test[i] = static_cast<uint32_t>(-1); */
  /*   data_test[i] = i; */
  /* } */

  /* test_prefix_sum<<<1, test_size>>>(data_test, return_test, test_size); */

  /* cuda_error_chk(hipDeviceSynchronize()); */

  /* for (size_t i = 0; i < test_size; ++i) { */
  /*   printf("i: %lu, r: %u\n", i, return_test[i]); */
  /* } */

  //assumptions:
  // - values per warp < 256
  // - num_warps_per_n is less than the warp_size

  uint32_t num_threads_per_block = 128;
  uint32_t num_warps_per_n =
      std::min((num_threads_per_block + warp_size - 1) / warp_size,
               ((m + warp_size - 1) / warp_size));

  size_t shared_count = (n * num_warps_per_n * BINS_PER_PASS) / 4 +
                        n * num_warps_per_n * BINS_PER_PASS + n * m * 4 +
                        BINS_PER_PASS * n;

  maxes<<<n, num_threads_per_block, shared_count * sizeof(int32_t)>>>(
      data, max_vals, max_locs, n, m, nth_max, n * num_threads_per_block,
      num_warps_per_n);

  cuda_error_chk(hipDeviceSynchronize());

  std::cout << "==== gpu ====\n";
  for (size_t i = 0; i < n; i++) {
    std::cout << "value: " << max_vals[i][0] << " loc: " << max_locs[i][0]
              << " value nth: " << max_vals[i][1]
              << " loc nth: " << max_locs[i][1] << "\n";
  }
  std::cout << std::endl;

  // print results;
  hipDeviceSynchronize();

  hipFree(data[0]);
  hipFree(data);

  return 0;
}
