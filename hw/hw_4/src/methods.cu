#include "hip/hip_runtime.h"
#include "methods.h"
#include "utils.h"
#include <algorithm>

void CRSMethodCPU::run() {
  for (int i = 0; i < Nrow; i++) {
    const int J1 = IA[i];
    const int J2 = IA[i + 1];
    double sum = 0.0;
    for (int j = 0; j < (J2 - J1); j++)
      sum += AA[j + J1] * x[JA[j + J1]];
    y[i] = sum;
  }
}

__global__ void SpMv_gpu_thread_CRS(int Nrow, double *AA, int *IA, int *JA,
                                    double *x, double *y) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < Nrow) {

    const int J1 = IA[i];
    const int J2 = IA[i + 1];
    double sum = 0.0;
    for (int j = 0; j < (J2 - J1); j++)
      sum += AA[j + J1] * x[JA[j + J1]];
    y[i] = sum;
  }
}

void CRSMethodGPU::run() {
  int num_threads = 64;
  SpMv_gpu_thread_CRS<<<(Nrow + num_threads - 1) / num_threads, num_threads>>>(
      Nrow, AA, IA, JA, x, y);
}

void CudaSparse::run() {
  double alpha = 1.;
  double beta = 0.;
  cuda_sparse_error_chk(hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       Nrow, Ncol, nnz, &alpha, descr, AA, IA,
                                       JA, x, &beta, y));
}

CudaSparse::CudaSparse(hipsparseHandle_t handle, int Nrow, int Ncol, int nnz,
                       double *AA, int *IA, int *JA, double *x, double *y)
    : CRSMethod(Nrow, AA, IA, JA, x, y), handle(handle), Ncol(Ncol), nnz(nnz) {
  cuda_sparse_error_chk(hipsparseCreateMatDescr(&descr));
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
}

void ELLPACKMethodCPU::run() {
  const int unroll_num = 4;
  #pragma omp parallel for
  for (int i = 0; i <  (Nrow / unroll_num) * unroll_num ; i += unroll_num) {
    double sum[unroll_num] = {0};

    int unroll_maxnzr = row_lengths[i];
    for (int k = 1; k < unroll_num; ++k) {
      unroll_maxnzr = std::max(unroll_maxnzr, row_lengths[i + k]);
    }
    for (int j = 0; j < unroll_maxnzr; j++) {
    #pragma unroll unroll_num
      for (int k = 0; k < unroll_num; k++) {
        sum[k] += AS[j][i + k] * x[JA[j][i + k]];
      }
    }
    #pragma unroll unroll_num
    for (int k = 0; k < unroll_num; k++) {
      y[i + k] = sum[k];
    }
  }
  for (int i = Nrow - Nrow % unroll_num; i < Nrow; i++) {
    double sum = 0;
    for (int j = 0; j < row_lengths[i]; j++) {
      sum += AS[j][i] * x[JA[j][i]];
    }
    y[i] = sum;
  }
}

static __inline__ __device__ double fetch_double(uint2 p) {
  return __hiloint2double(p.y, p.x);
}

__device__ double SpMv_gpu_thread_ELLPACK_row(int row_length, double **AS,
                                              int **JA, hipTextureObject_t x,
                                              int row) {
  double sum = 0;
  for (int j = 0; j < row_length; j++) {
    sum += AS[j][row] * fetch_double(tex1Dfetch<uint2>(x, JA[j][row]));

  }
  return sum;
}

__global__ void SpMv_gpu_thread_ELLPACK(int Nrow, int maxnzr, int *row_lengths,
                                        double **AS, int **JA,
                                        hipTextureObject_t x, double *y) {
  // compute y = A*x
  // A is sparse operator stored in a ELLPACK format

  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < Nrow) {
    y[row] =
        SpMv_gpu_thread_ELLPACK_row(row_lengths[row], AS, JA, x, row);
  }
}

void ELLPACKMethodGPU::run() {
  int num_threads = 64;

  SpMv_gpu_thread_ELLPACK<<<(Nrow + num_threads - 1) / num_threads,
                            num_threads>>>(Nrow, maxnzr, row_lengths, AS, JA, x,
                                           y);
}
