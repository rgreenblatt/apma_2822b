#include "cuda_utils.hpp"

namespace miniFE {
void select_cuda_device(int mpi_rank) {
  int device_count;
  hipGetDeviceCount(&device_count);
  hipSetDevice(mpi_rank % device_count);
}
} // namespace miniFE
