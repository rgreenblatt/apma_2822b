#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "vector_uvm.hpp"

template <class T> T *UMAllocator<T>::allocate(size_t n) {
  T *ptr;
#ifdef USE_CUDA
  if (n > 0) {
    //        ptr = (T*) malloc(n*sizeof(T));
    //        hipMemPrefetchAsync(ptr,n*sizeof(T),0,0);
    hipMallocManaged(&ptr, n * sizeof(T));
    hipMemAdvise(ptr, n * sizeof(T), hipMemAdviseSetPreferredLocation, 0);
  } else {
    ptr = NULL;
  }
#else
  ptr = new T[n];
#endif
  return ptr;
}

template <class T> void UMAllocator<T>::deallocate(T *p, size_t) {
#ifdef USE_CUDA
  hipFree(p);
#else
  delete p;
#endif
}

template struct UMAllocator<MINIFE_LOCAL_ORDINAL>;
template struct UMAllocator<MINIFE_GLOBAL_ORDINAL>;
template struct UMAllocator<MINIFE_SCALAR>;
