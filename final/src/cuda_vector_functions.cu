#include "hip/hip_runtime.h"
#include "cuda_vector_functions.hpp"
#include "waxpby_op.cuh"
#include "cuda_utils.cuh"

template <typename ScalarType>
__global__ void cuda_waxpby_fused_kernel(
    ScalarType *wcoefs, ScalarType *w2coefs, ScalarType alpha,
    ScalarType alpha2, const ScalarType *xcoefs, const ScalarType *x2coefs,
    ScalarType beta, ScalarType beta2, const ScalarType *ycoefs,
    const ScalarType *y2coefs, size_t n) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    waxpby_op(wcoefs, alpha, xcoefs, beta, ycoefs, i);
    waxpby_op(w2coefs, alpha2, x2coefs, beta2, y2coefs, i);
  }
}

template <typename ScalarType>
void cuda_waxpby_fused(ScalarType *wcoefs, ScalarType *w2coefs, ScalarType alpha,
                  ScalarType alpha2, const ScalarType *xcoefs,
                  const ScalarType *x2coefs, ScalarType beta, ScalarType beta2,
                  const ScalarType *ycoefs, const ScalarType *y2coefs,
                  unsigned n) {

  const unsigned thread_num = 256;
  cuda_waxpby_fused_kernel<<<(n + thread_num - 1) / thread_num, thread_num>>>(wcoefs, w2coefs, alpha, alpha2, xcoefs, x2coefs, beta, beta2, ycoefs,
      y2coefs, n);
  cuda_error_chk(hipDeviceSynchronize());
}

template void cuda_waxpby_fused(MINIFE_SCALAR *wcoefs, MINIFE_SCALAR *w2coefs,
                                MINIFE_SCALAR alpha, MINIFE_SCALAR alpha2,
                                const MINIFE_SCALAR *xcoefs,
                                const MINIFE_SCALAR *x2coefs,
                                MINIFE_SCALAR beta, MINIFE_SCALAR beta2,
                                const MINIFE_SCALAR *ycoefs,
                                const MINIFE_SCALAR *y2coefs, unsigned n);
